
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>
#include<ctime>
#include<cstdlib>

__global__ void add(int *a, int *b, int *c)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    c[index] = a[index] + b[index];
}

int main(){
    int size = 1e9; 
    
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int *h_a, *h_b, *h_c;
    int *h_sum;
    int *d_sum;
    int totalSize = size * sizeof(int);
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    a = (int *)malloc(totalSize);
    b = (int *)malloc(totalSize);
    c = (int *)malloc(totalSize);
    h_sum = (int *)malloc(totalSize);

    hipMalloc(&d_a, totalSize);
    hipMalloc(&d_b, totalSize);
    hipMalloc(&d_c, totalSize);
    hipMalloc(&d_sum, totalSize);

    h_a = a;
    h_b = b;
    h_c = c;

    srand(time(0));
    for(int i = 0; i < size; i++){
        h_a[i] = rand() % 100;
        h_b[i] = rand() % 100;
    }

    hipMemcpy(d_a, h_a, totalSize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, totalSize, hipMemcpyHostToDevice);

    int start_time = clock();
    add<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c);
    int end_time = clock();

    hipMemcpy(h_c, d_c, totalSize, hipMemcpyDeviceToHost);

    for(int i = 0; i < size; i++){
        h_sum[i] = h_a[i] + h_b[i];
    }

    hipMemcpy(d_sum, h_sum, totalSize, hipMemcpyHostToDevice);

    for(int i = 0; i < size; i++){
        if(h_c[i] != h_sum[i]){
            printf("Error at index %d\n", i);
            break;
        }
    }

    free(a);
    free(b);
    free(c);
    free(h_sum);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_sum);

    printf("Time taken: %f\n", (end_time - start_time) / (double) CLOCKS_PER_SEC);
    return 0;
}

// nvcc -o w14_cuda w14_cuda.cu && ./w14_cuda