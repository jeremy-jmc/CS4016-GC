

#include <hip/hip_runtime.h>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <hiprand/hiprand_kernel.h>

struct Matrix {
  int rows;
  int cols;
  float *M;
};

__global__ void populateMatrixKernel(float *M, int rows, int cols, unsigned long seed) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int totalSize = rows * cols;
  if (idx < totalSize) {
    hiprandState state;
    hiprand_init(seed, idx, 0, &state);
    M[idx] = hiprand_uniform(&state);
  }
}

void populateMatrix(Matrix &M1) {
  int totalSize = M1.rows * M1.cols;
  float *d_M;
  hipMalloc(&d_M, totalSize * sizeof(float));
  int threadsPerBlock = 256;
  int blocksPerGrid = (totalSize + threadsPerBlock - 1) / threadsPerBlock;

  populateMatrixKernel<<<blocksPerGrid, threadsPerBlock>>>(d_M, M1.rows, M1.cols, time(0));

  hipMemcpy(M1.M, d_M, totalSize * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(d_M);
}

__global__ void multiplyMatricesKernel(float *M1, int rows1, int cols1, float *M2, int rows2, int cols2, float *M3) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < rows1 && col < cols2) {
    float sum = 0;
    for (int k = 0; k < cols1; ++k) {
      sum += M1[row * cols1 + k] * M2[k * cols2 + col];
    }
    M3[row * cols2 + col] = sum;
  }
}

void multiplyMatrices(Matrix &M1, Matrix &M2, Matrix &M3) {
  M3.rows = M1.rows;
  M3.cols = M2.cols;
  int sizeM1 = M1.rows * M1.cols * sizeof(float);
  int sizeM2 = M2.rows * M2.cols * sizeof(float);
  int sizeM3 = M3.rows * M3.cols * sizeof(float);

  float *d_M1, *d_M2, *d_M3;
  hipMalloc(&d_M1, sizeM1);
  hipMalloc(&d_M2, sizeM2);
  hipMalloc(&d_M3, sizeM3);

  hipMemcpy(d_M1, M1.M, sizeM1, hipMemcpyHostToDevice);
  hipMemcpy(d_M2, M2.M, sizeM2, hipMemcpyHostToDevice);

  dim3 threadsPerBlock(16, 16);
  dim3 blocksPerGrid((M3.cols + threadsPerBlock.x - 1) / threadsPerBlock.x,
                     (M3.rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

  multiplyMatricesKernel<<<blocksPerGrid, threadsPerBlock>>>(d_M1, M1.rows, M1.cols, d_M2, M2.rows, M2.cols, d_M3);

  hipMemcpy(M3.M, d_M3, sizeM3, hipMemcpyDeviceToHost);

  hipFree(d_M1);
  hipFree(d_M2);
  hipFree(d_M3);
}

void printMatrix(Matrix M1) {
  for (int i = 0; i < M1.rows; ++i) {
    for (int j = 0; j < M1.cols; ++j) {
      std::cout << M1.M[i * M1.cols + j] << " ";
    }
    std::cout << std::endl;
  }
}

int main() {
  srand(static_cast<unsigned>(time(0)));

  int widthA, heightA, widthB, heightB;

  std::cout << "Enter the dimensions of matrix A (width height): ";
  std::cin >> widthA >> heightA;

  std::cout << "Enter the dimensions of matrix B (width height): ";
  std::cin >> widthB >> heightB;

  if (widthA != heightB) {
    std::cerr << "Error: Incompatible dimensions for matrix multiplication."
              << std::endl;
    return -1;
  }

  Matrix M1 = {heightA, widthA, new float[heightA * widthA]};
  Matrix M2 = {heightB, widthB, new float[heightB * widthB]};
  Matrix M3 = {heightA, widthB, new float[heightA * widthB]};

  populateMatrix(M1);
  populateMatrix(M2);

  std::cout << "Matrix A:" << std::endl;
  //printMatrix(M1);

  std::cout << "Matrix B:" << std::endl;
  //printMatrix(M2);

  multiplyMatrices(M1, M2, M3);
  std::cout << "Resulting Matrix C computed" << std::endl;
  //printMatrix(M3);

  delete[] M1.M;
  delete[] M2.M;
  delete[] M3.M;

  return 0;
}

// nvcc -o w14_matmul w14_matmul.cu && ./w14_matmul